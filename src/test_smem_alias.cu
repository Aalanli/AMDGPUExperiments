
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>


__device__ __forceinline__ void test(char* a) {
    __shared__ char s[0xc000];
    int tid = threadIdx.x;
    s[tid] = a[tid];
    a[tid] = s[(tid + 1) % blockDim.x];
}

__global__ void alias_test(char *a, char *b) {
    __shared__ char s[0xc000];
    int tid = threadIdx.x;
    s[tid] = a[tid];
    a[tid] = s[(tid + 1) % blockDim.x];
    __syncthreads();
    test(b);
}


int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Shared memory per block: %zu\n", prop.sharedMemPerBlock);

    char *a, *b;
    hipMalloc(&a, 49152 * sizeof(char));
    hipMalloc(&b, 49152 * sizeof(char));

    alias_test<<<1, 1024>>>(a, b);
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    // printf("%d", sizeof(short));

}