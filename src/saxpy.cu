
#include <hip/hip_runtime.h>
#include <stdio.h>

#ifndef LAUNCH_NAME
#define LAUNCH_NAME saxpy
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 256
#endif

#ifndef REPEATS
#define REPEATS 4
#endif

__global__ void saxpy_kernel(
    const float* __restrict__ a,
    const float* __restrict__ b,
    float* __restrict__ c,
    int n, int d) 
{
    int bid = blockIdx.y;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid < d) {
        c[bid * d + tid] = a[bid * d + tid] + b[bid * d + tid];
        tid += stride;
    }
}

extern "C" bool LAUNCH_NAME(float* a, float* b, float* c, int n, int d);

bool LAUNCH_NAME(float* a, float* b, float* c, int n, int d) {
    const int repeats = BLOCKSIZE * REPEATS;
    saxpy_kernel<<<dim3((d + repeats - 1) / repeats, n), BLOCKSIZE>>>(a, b, c, n, d);
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return false;
    }

    return true;
}

