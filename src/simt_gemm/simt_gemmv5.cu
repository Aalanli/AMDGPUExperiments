#include "hip/hip_runtime.h"
#include <cstdio>
#include "utils.hpp"

#ifndef LAUNCH_NAME
#define LAUNCH_NAME simt_gemm
#define IS_EXE
#endif

#ifndef TYPE
#define TYPE float
#endif

#ifndef BlockM
#define BlockM 16
#endif

#ifndef BlockK
#define BlockK 32
#endif

#ifndef BlockN
#define BlockN 64
#endif

#ifndef WarpM
#define WarpM 2
#endif

#ifndef WarpN
#define WarpN 1
#endif

#ifndef ThreadM
#define ThreadM 2
#endif

#ifndef ThreadK
#define ThreadK 1
#endif

#ifndef ThreadN
#define ThreadN 16
#endif

#ifndef TM
#define TM 4
#endif

#ifndef TK
#define TK 1
#endif

#ifndef TN
#define TN 4
#endif

__device__ __host__ constexpr int load_factor(const int nthreads, const int min_contiguous, const int dim) {
    int max_load_factor = dim / min_contiguous;
    while (max_load_factor > 1) {
        if (nthreads % (max_load_factor * min_contiguous) == 0) {
            return max_load_factor;
        }
        max_load_factor--;
    }
    return 1;
}

__device__ __host__ constexpr int calc_min_contiguous(const int dim, const int warp_size) {
    int min_contiguous = dim % warp_size == 0 ? warp_size : warp_size / 2;
    min_contiguous = dim % min_contiguous == 0 ? min_contiguous : min_contiguous / 2;
    min_contiguous = dim % min_contiguous == 0 ? min_contiguous : min_contiguous / 2;    
    min_contiguous = dim % min_contiguous == 0 ? min_contiguous : min_contiguous / 2;    
    return min_contiguous;
}


/// this function must be hit by all threads
/// LoadF: (int, int) -> T
/// StoreF: (int, int, T) -> void
template <int NThreads, int WarpSize, typename T, int D1, int D2, typename LoadF, typename StoreF>
__device__ __forceinline__ void coalesce_mem_2d(LoadF&& load_f, StoreF&& store_f) {
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    // quarter warp should still be fine for global loads
    constexpr int min_contiguous = calc_min_contiguous(D2, WarpSize);
    static_assert(D2 % min_contiguous == 0, "last dimension is not contiguous enough");
    /// find how many threads to factor along the last dimension
    constexpr int factor = load_factor(NThreads, min_contiguous, D2);
    constexpr int factor_threads = factor * min_contiguous;
    constexpr int stride = NThreads / factor_threads;
    for (int i = 0; i < D1; i += stride) {
        for (int j = 0; j < D2; j += factor_threads) {
            int tid_contiguous = tid % factor_threads;
            int tid_factor = tid / factor_threads;
            int coord_i = i + tid_factor;
            int coord_j = j + tid_contiguous;
            if (D1 % stride == 0 && stride < D1) {
                store_f(coord_i, coord_j, load_f(coord_i, coord_j));
            } else {
                if (coord_i < D1) { // coord_j is always in bounds
                    store_f(coord_i, coord_j, load_f(coord_i, coord_j));
                }
            }
        }
    }
}

/// F is a function of type (int, int) -> T
template <int NThreads, int WarpSize, typename T, int D1, int D2, typename F>
__device__ __forceinline__ void load_smem(Tensor<T, D1, D2> &a, F&& f) {
    coalesce_mem_2d<NThreads, WarpSize, T, D1, D2>(f, [&](int i, int j, T v) {
        a[i][j] = v;
    });
}

/// F is a function of type (int, int) -> T
template <int NThreads, int WarpSize, typename T, int D1, int D2, typename F>
__device__ __forceinline__ void store_gmem(Tensor<T, D1, D2> &a, F&& f) {
    coalesce_mem_2d<NThreads, WarpSize, T, D1, D2>([&](int i, int j) {
        return a[i][j];
    }, f);
}


template <int M, int K, int N>
__device__ __forceinline__ void mma(float (&a)[M][K], float (&b)[K][N], float (&c)[M][N]) {
    #pragma unroll
    for (int m = 0; m < M; ++m) {
        #pragma unroll
        for (int n = 0; n < N; ++n) {
            #pragma unroll
            for (int k = 0; k < K; ++k) {
                c[m][n] = c[m][n] + a[m][k] * b[k][n];
            }
        }
    }
}

template <int warpSize>
__device__ __forceinline__ float warp_reduce(float data) {
    int lane = threadIdx.x % warpSize;
    for (int i = 1; i < warpSize; i *= 2)
        data += __shfl_xor(data, i);
    return data;
}


template <int D1, int D2>
__device__ void debug_print_smem(Tensor<float, D1, D2> &a) {
    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
        for (int i = 0; i < D1; ++i) {
            for (int j = 0; j < D2; ++j) {
                printf("%f ", a[i][j]);
            }
            printf("\n");
        }
        printf("\n");
    }
}

// 1. effect of vectorized load
// 2. effect of block-level smem pipeling (global -> reg -> smem)
// 3. effect of warp-level reg pipeling (smem -> reg)
template <int warpSize>
__global__ void __launch_bounds__(WarpM * /*WarpK* */ WarpN * warpSize) simt_gemm_kernelv5(
    const float* __restrict__ a, 
    const float* __restrict__ b, 
    float* __restrict__ c,
    const int M, const int N, const int K
) {
    constexpr int WarpK = 1; // if WarpK > 1, then we have to reduce in smem

    constexpr int nthreads = WarpM * WarpK * WarpN * warpSize;
    // stage 1. each block handles A[BlockM, BlockK] x B[BlockK, BlockN]
    // stage 2. each warp handles A[ThreadM, BlockK / WarpK] x B[BlockK / WarpK, ThreadN]
    // stage 3. each thread handles A[TM, TK] x B[TK, TN]
    constexpr int smem_elems = Max<2 * (BlockM * (BlockK) + BlockK * (BlockN)), 0>::value;
    __shared__ float smem[smem_elems];

    static_assert(BlockK >= WarpK * ThreadK * TK,"");
    
    static_assert(BlockM >= WarpM * ThreadM * TM && BlockM % (WarpM * ThreadM * TM) == 0, "");
    static_assert(BlockN >= WarpN * ThreadN * TN && BlockN % (WarpN * ThreadN * TN) == 0, "");
    static_assert(next_power_of_2(ThreadK) == ThreadK, "");
    constexpr int WM_REP = BlockM / (WarpM * ThreadM * TM);
    constexpr int WN_REP = BlockN / (WarpN * ThreadN * TN);
    float regs_c[WM_REP][WN_REP][TM][TN];
    for (int wmr = 0; wmr < WM_REP; ++wmr) {
        for (int wnr = 0; wnr < WN_REP; ++wnr) {
            for (int tm = 0; tm < TM; ++tm) {
                for (int tn = 0; tn < TN; ++tn) {
                    regs_c[wmr][wnr][tm][tn] = 0.0f;
                }
            }
        }
    }
    
    Tensor<float, 2, BlockK, BlockM> sA(smem);
    Tensor<float, 2, BlockK, BlockN> sB(smem + 2 * (BlockM) * (BlockK));
    // float* sA = smem;
    // float* sB = smem + BlockM * BlockK;
    static_assert(nthreads >= BlockK, "");
    static_assert(nthreads % BlockK == 0, "");
    constexpr int stride_m = nthreads / BlockK;
    constexpr int repeat_m = BlockM / stride_m;
    static_assert(repeat_m > 0, "");
    
    auto load_a = [&](int kb, int smem_tile) {
            int klane = threadIdx.x % BlockK;
            int mrow = threadIdx.x / BlockK;
            if (repeat_m % 4 == 0) { // shared store can optimize to 128 bytes, so no 4 way bank conflict 
                #pragma unroll
                for (int i = mrow * 4; i < BlockM; i += stride_m * 4) {
                    float buf[4];
                    for (int j = 0; j < 4; ++j) {
                        int coord_m = (blockIdx.x * BlockM) + i + j;
                        int coord_k = kb * BlockK + klane;
                        bool inbounds = coord_m < M && coord_k < K;
                        buf[j] = inbounds ? a[coord_m * K + coord_k] : 0;
                    }
                    for (int j = 0; j < 4; ++j) {
                        sA[smem_tile][klane][i + j + (klane * 4) % BlockM] = buf[j];
                    }
                }
            } else {
                float buf[repeat_m];
                for (int i = mrow; i < BlockM; i += stride_m) {
                    int coord_m = (blockIdx.x * BlockM) + i;
                    int coord_k = kb * BlockK + klane;
                    bool inbounds = coord_m < M && coord_k < K;
                    buf[i] = inbounds ? a[coord_m * K + coord_k] : 0;
                }
                for (int i = mrow; i < BlockM; i += stride_m) {
                    sA[smem_tile][klane][i] = buf[i];
                }
            }
        };

    static_assert(nthreads >= BlockN, "");
    static_assert(nthreads % BlockN == 0, "");
    constexpr int stride_k = nthreads / BlockN;
    constexpr int repeat_k = BlockN / stride_k;
    static_assert(repeat_k > 0, "");
    auto load_b = [&](int kb, int smem_tile) {
            int nlane = threadIdx.x % BlockN;
            int krow = threadIdx.x / BlockN;
            if (repeat_k % 4 == 0) {
                #pragma unroll
                for (int i = krow * 4; i < BlockK; i += stride_k * 4) {
                    float buf[4];
                    for (int j = 0; j < 4; ++j) {
                        int coord_k = kb * BlockK + i + j;
                        int coord_n = blockIdx.y * BlockN + nlane;
                        bool inbounds = coord_k < K && coord_n < N;
                        buf[j] = inbounds ? b[coord_k * N + coord_n] : 0;
                    }
                    for (int j = 0; j < 4; ++j) {
                        sB[smem_tile][i + j][nlane] = buf[j];
                    }
                }
            } else {
                float buf[repeat_k];
                for (int i = krow; i < BlockK; i += stride_k) {
                    int coord_k = kb * BlockK + i;
                    int coord_n = blockIdx.y * BlockN + nlane;
                    bool inbounds = coord_k < K && coord_n < N;
                    buf[i] = inbounds ? b[coord_k * N + coord_n] : 0;
                }
                for (int i = krow; i < BlockK; i += stride_k) {
                    sB[smem_tile][i][nlane] = buf[i];
                }
            }
        };
    
    

    float regs_a[2][WM_REP][WN_REP][TM][TK];
    float regs_b[2][WM_REP][WN_REP][TK][TN];
    int warp_id = threadIdx.x / warpSize;
    int warp_k_offset = (warp_id % WarpK) * ThreadK;
    int warp_m_offset = ((warp_id / WarpK) % WarpM) * ThreadM;
    int warp_n_offset = ((warp_id / WarpK) / WarpM) * ThreadN;
    auto load_regs_a = [&](int wk, int smem_tile, int regs_tile) {
        for (int wm = 0; wm < WM_REP; ++wm) {
            for (int wn = 0; wn < WN_REP; ++wn) {
                int lane = threadIdx.x % warpSize;
                int thread_m_offset = ((lane / ThreadK) % ThreadM + warp_m_offset + wm * WarpM * ThreadM) * TM;

                #pragma unroll
                for (int ik = 0; ik < TK; ik++) {
                    #pragma unroll
                    for (int im = 0; im < TM; im++) {
                        int thread_k_offset = (lane % ThreadK + warp_k_offset) * TK + wk;
                        int coord_k = thread_k_offset + ik;
                        if (repeat_m % 4 == 0)
                            regs_a[regs_tile][wm][wn][im][ik] = sA[smem_tile][coord_k][(thread_m_offset + im) + (coord_k * 4) % BlockM];
                        else
                            regs_a[regs_tile][wm][wn][im][ik] = sA[smem_tile][coord_k][thread_m_offset + im];
                    }
                }
            }
        }
    };
    auto load_regs_b = [&](int wk, int smem_tile, int regs_tile) {
        for (int wm = 0; wm < WM_REP; ++wm) {
            for (int wn = 0; wn < WN_REP; ++wn) {
                int lane = threadIdx.x % warpSize;
                int thread_n_offset = ((lane / ThreadK / ThreadM) + warp_n_offset + wn * WarpN * ThreadN) * TN;

                int thread_k_offset = (lane % ThreadK + warp_k_offset) * TK + wk;
                #pragma unroll
                for (int ik = 0; ik < TK; ik++) {
                    #pragma unroll
                    for (int in = 0; in < TN; in++) {
                        regs_b[regs_tile][wm][wn][ik][in] = sB[smem_tile][(thread_k_offset + ik)][thread_n_offset + in];
                    }
                }
            }
        }
    };
    auto mma_ = [&](int wk) {
        for (int wm = 0; wm < WM_REP; ++wm)
            for (int wn = 0; wn < WN_REP; ++wn)
                mma(regs_a[wk % 2][wm][wn], regs_b[wk % 2][wm][wn], regs_c[wm][wn]);
    };

    load_a(0, 0);
    load_b(0, 0);
    __syncthreads();
    constexpr int reps_wk = BlockK / (WarpK * ThreadK * TK);
    int k_tiles = (K + BlockK - 1) / BlockK - 1;
    for (int kb = 0; kb < k_tiles; kb++) {
        load_regs_a(0, kb % 2, 0);
        load_regs_b(0, kb % 2, 0);
        #pragma unroll
        for (int wk = 0; wk < reps_wk - 1; ++wk) {
            mma_(wk);            
            load_regs_a((wk + 1) * WarpK * ThreadK * TK, kb % 2, (wk + 1) % 2);
            load_regs_b((wk + 1) * WarpK * ThreadK * TK, kb % 2, (wk + 1) % 2);
        }
        mma_((reps_wk - 1));    
        
        load_a(kb + 1, (kb + 1) % 2);
        load_b(kb + 1, (kb + 1) % 2);
        __syncthreads();
    }
    load_regs_a(0, k_tiles % 2, 0);
    load_regs_b(0, k_tiles % 2, 0);
    #pragma unroll
    for (int wk = 0; wk < reps_wk - 1; ++wk) {
        mma_(wk);            
        load_regs_a((wk + 1) * WarpK * ThreadK * TK, k_tiles % 2, (wk + 1) % 2);
        load_regs_b((wk + 1) * WarpK * ThreadK * TK, k_tiles % 2, (wk + 1) % 2);
    }
    mma_((reps_wk - 1));


    {
        int warp_id = threadIdx.x / warpSize;
        int warp_k_offset = (warp_id % WarpK) * ThreadK;
        int warp_m_offset = ((warp_id / WarpK) % WarpM) * ThreadM;
        int warp_n_offset = ((warp_id / WarpK) / WarpM) * ThreadN;
        for (int wm = 0; wm < WM_REP; ++wm) {
            for (int wn = 0; wn < WN_REP; ++wn) {
                int lane = threadIdx.x % warpSize;
                if (lane % ThreadK == 0) {
                    int thread_m_offset = ((lane / ThreadK) % ThreadM + warp_m_offset + wm * WarpM * ThreadM) * TM;
                    int thread_n_offset = ((lane / ThreadK / ThreadM) + warp_n_offset + wn * WarpN * ThreadN) * TN;
                    
                    for (int im = 0; im < TM; im++) {
                        for (int in = 0; in < TN; in++) {
                            // sC[thread_m_offset + im][thread_n_offset + in] = regs_c[wm][wn][im][in];
                            int coord_m = blockIdx.x * BlockM + thread_m_offset + im;
                            int coord_n = blockIdx.y * BlockN + thread_n_offset + in;
                            if (coord_m < M && coord_n < N) {
                                c[coord_m * N + coord_n] = regs_c[wm][wn][im][in];
                            }
                        }
                    }
                }
            }
        }
    }
}

EXPORT bool LAUNCH_NAME(float* a, float* b, float* c, int m, int k, int n) {

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int warp_size = prop.warpSize;
    int smem = prop.sharedMemPerBlock;
    int regs = prop.regsPerBlock;

    dim3 grid(cdiv(m, BlockM), cdiv(n, BlockN));
    dim3 block(WarpM * WarpN * warp_size);

    if (ThreadM * ThreadK * ThreadN != warp_size) {
        printf("ThreadM * ThreadK * ThreadN != warp_size, (%d, %d, %d) != %d\n", ThreadM, ThreadK, ThreadN, warp_size);
        return false;
    }

    auto kernel = simt_gemm_kernelv5<32>; 
    kernel<<<grid, block>>>(a, b, c, m, n, k);

    

    // check error
    auto error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        return false;
    }

    return true;
}

#ifdef IS_EXE
int main() {
    float *a, *b, *c;
    hipMalloc(&a, 1024 * 1024 * sizeof(float));
    hipMalloc(&b, 1024 * 1024 * sizeof(float));
    hipMalloc(&c, 1024 * 1024 * sizeof(float));
    LAUNCH_NAME(a, b, c, 1024, 1024, 1024);
    hipFree(a);
    hipFree(b);
    hipFree(c); 
}
#endif